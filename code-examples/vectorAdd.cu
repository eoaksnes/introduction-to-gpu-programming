#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(float *d_A, float *d_B, float *d_C, int n) {
    //int i = blockDim.x * blockIdx.x + threadIdx.x;
    int i = threadIdx.x;
    // Make sure we do not go out of bounds.   
    if (i < n) {
        d_C[i] = d_A[i] + d_B[i];
    }
}

int main() {
    // Size of vectors
    int n = 16;

    // Size, in bytes, of each vector
    size_t bytes = n * sizeof(float);

    // Allocate memory for the host vectors
    float *h_A = (float *)malloc(bytes);
    float *h_B = (float *)malloc(bytes);
    float *h_C = (float *)malloc(bytes);
    
    // Initialize vectors
    for (int i = 0; i < n; ++i) {
      h_A[i] = 1; 
      h_B[i] = 2;
    }

    // Allocate memory for the device vectors
    float *d_A = NULL;
    hipMalloc((void **)&d_A, bytes);
    float *d_B = NULL;
    hipMalloc((void **)&d_B, bytes);
    float *d_C = NULL;
    hipMalloc((void **)&d_C, bytes);
    
    // Copy the host vectors to the device    
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    // Execute the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(n + threadsPerBlock - 1) / threadsPerBlock;
    //vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, n);

    vectorAdd<<<1, n>>>(d_A, d_B, d_C, n);

    printf("Kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    // Copy the result back to the host result vector
    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    // Sum up host result vector and print result divided by n, this should equal 4
    float sum = 0;
    int i;
    for(i=0; i<n; i++) {
        sum += h_C[i];
    }
    sum = sum/n;
    printf("Final result: %f\n", sum);

    // Free device global memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Explicitly destroys and cleans up all resources associated with the current device
    hipDeviceReset();
    return 0;
}